#include "hip/hip_runtime.h"
#include "GPUSolver.cuh"
#include "MyParameterization.h"
#include <boost/numeric/ublas/io.hpp>
#include <boost/numeric/ublas/triangular.hpp>
#include <boost/numeric/ublas/vector.hpp>
#include <boost/numeric/ublas/matrix_sparse.hpp>
#include <boost/numeric/ublas/matrix.hpp>
#include <boost/numeric/ublas/matrix_proxy.hpp>
#include <boost/numeric/ublas/operation.hpp>
#include <boost/numeric/ublas/operation_sparse.hpp>
#include <boost/numeric/ublas/io.hpp>
#include <boost/numeric/ublas/lu.hpp>

//#include "viennacl/scalar.hpp"
#include "viennacl/vector.hpp"
#include "viennacl/compressed_matrix.hpp"
#include "viennacl/matrix.hpp"
#include "viennacl/matrix_proxy.hpp"
#include "viennacl/linalg/bicgstab.hpp"
#include "viennacl/linalg/cg.hpp"
#include "viennacl/linalg/gmres.hpp"
#include "viennacl/linalg/direct_solve.hpp"
#include "viennacl/linalg/lu.hpp"
#include "viennacl/linalg/ichol.hpp"

//Parallel Patterns Library
#include <ppl.h>
#include <concrt.h>


typedef double ScalarType;
//typedef boost::numeric::ublas::compressed_matrix<ScalarType>        cpuCompressedMatrixType;
typedef boost::numeric::ublas::compressed_matrix<ScalarType>        cpuCompressedMatrixType;
typedef boost::numeric::ublas::matrix<ScalarType>					cpuDenseMatrixType;
typedef boost::numeric::ublas::vector<ScalarType>					cpuVectorType;
typedef viennacl::compressed_matrix<ScalarType>						gpuCompressedMatrixType;
typedef viennacl::matrix<ScalarType>								gpuDenseMatrixType;
typedef viennacl::vector<ScalarType>								gpuVectorType;

using namespace boost::numeric;
void MyParameterization::ParametrizationOptimalGPU(double error,FILE* logFile)
{

	//int numberV = 100;
	cpuCompressedMatrixType cpuAmatrix(numberV*2,numberV*2);
	cpuVectorType cpuBvector(numberV*2);
	cpuVectorType cpuXvector(numberV*2);
	
	gpuCompressedMatrixType gpuAmatrix(numberV*2,numberV*2);
	gpuVectorType gpuBvector(numberV*2);
	gpuVectorType gpuXvector(numberV*2);
	
	setFloaterC();
	SortIndexP();

	PolarList *nowp = NULL;
	for (int i = 0; i < numberV ; i++)
	{
		if(boundary[i]!=1)
		{
			cpuBvector(i) = 0.0f;
			cpuBvector(i+numberV) = 0.0f;

			nowp = PHead[i];
			
			cpuAmatrix(i,i) = 1.0f;
			cpuAmatrix(numberV +i,numberV +i) = 1.0f;
			while(nextN(nowp)!=PTail[i])
			{
				nowp = nextN(nowp);						
				cpuAmatrix(i,nowp->ID) = -nowp->lambda;		
				cpuAmatrix(numberV +i,numberV + nowp->ID) = -nowp->lambda;
			}			
		}
		else
		{			
			//constraint 
			cpuBvector(i) = pU[i];
			cpuBvector(i+numberV) = pV[i];

			cpuAmatrix(i,i) = 1.0f;
			cpuAmatrix(numberV + i,numberV +i) = 1.0f;
		}
	}
			
	
	viennacl::copy(cpuAmatrix, gpuAmatrix);
	viennacl::copy(cpuBvector, gpuBvector);
	
			
	viennacl::linalg::ilu0_tag ilu0_config;
	viennacl::linalg::ilu0_precond< gpuCompressedMatrixType > vcl_ilut(gpuAmatrix,ilu0_config);
	clock_t calTime = 0;
	
	gpuXvector = viennacl::linalg::solve(gpuAmatrix,gpuBvector,viennacl::linalg::bicgstab_tag(error),vcl_ilut);
	calTime = clock() - calTime; 
	viennacl::copy(gpuXvector, cpuXvector);

	for(int i=0;i<numberV;i++)
	{
		if(boundary[i]!=1)
		{
			pU[i] = cpuXvector(i);
			pV[i] = cpuXvector(numberV + i);
					
		}
	}
	
	
	
	
	double previous_l2= 0;
	double candidate_l2 = this->getCurrentE();
	this->resultStretch = candidate_l2;
	double *sigsum = new double[numberV];
	double *prevU = new double[numberV];
	double *prevV = new double[numberV];
	do
	{
		previous_l2 = candidate_l2;
		memcpy(prevU,pU,sizeof(double)*numberV);
		memcpy(prevV,pV,sizeof(double)*numberV);
		setSigmaZero();
		for(int i=0;i<numberV;i++)
		{      
			if(boundary[i]!=1)
			{
				sigsum[i]=0.0;
				nowp = PHead[i];
				while(nextN(nowp)!=PTail[i])
				{
					nowp = nextN(nowp);
					nowp->old_lambda = nowp->lambda;
					nowp->lambda /= sigma[nowp->ID];
					sigsum[i] += nowp->lambda;
				}

				cpuAmatrix(i,i) = 1.0f;
				cpuAmatrix(i+numberV,i+numberV) = 1.0f;
				nowp = PHead[i];
				while(nextN(nowp)!=PTail[i])
				{
					nowp = nextN(nowp);
					cpuAmatrix(i,nowp->ID) = -nowp->lambda/sigsum[i];	
					cpuAmatrix(i+numberV,nowp->ID+numberV) = -nowp->lambda/sigsum[i];	
				}
			}
			else
			{
				cpuAmatrix(i,i) = 1.0f;
				cpuAmatrix(i+numberV,i+numberV) = 1.0f;
			}
		}

		viennacl::copy(cpuAmatrix, gpuAmatrix);
		viennacl::copy(cpuBvector, gpuBvector);

		viennacl::linalg::ilu0_tag ilu0_config;
		viennacl::linalg::ilu0_precond< gpuCompressedMatrixType > vcl_ilut(gpuAmatrix,ilu0_config);
		clock_t calTime = 0;
		calTime = clock();
		gpuXvector = viennacl::linalg::solve(gpuAmatrix,gpuBvector,viennacl::linalg::bicgstab_tag(error),vcl_ilut);
		calTime = clock() - calTime; 
		viennacl::copy(gpuXvector, cpuXvector);

		for(int i=0;i<numberV;i++)
		{
			if(boundary[i]!=1)
			{
				
				pU[i] = cpuXvector(i);
				pV[i] = cpuXvector(numberV + i);
					
			}
		}		
		candidate_l2 = this->getCurrentE();
	}
	while (previous_l2 > candidate_l2);

	memcpy(pU,prevU,sizeof(double)*numberV);
	memcpy(pV,prevV,sizeof(double)*numberV);

	delete [] prevU;
	delete [] prevV;
	this->resultStretch = previous_l2;
}


double MyParameterization::ParametrizationOptimalGPU(double *ioU,double *ioV,double error, cpuCompressedMatrixType *initAMatrix,bool directsolver,FILE* logFile)
{
	cpuCompressedMatrixType cpuAmatrix(numberV*2,numberV*2);
	if (initAMatrix != NULL && 
		initAMatrix->size1() == initAMatrix->size2() &&
		initAMatrix->size1() == numberV)
	{
		ublas::project(cpuAmatrix,ublas::range(0, numberV),ublas::range(0, numberV)) = *initAMatrix;
		ublas::project(cpuAmatrix,ublas::range(numberV, 2*numberV),ublas::range(numberV, 2*numberV)) = *initAMatrix;
		//ublas::subrange(cpuAmatrix,0,numberV,0,numberV) = *initAMatrix;
		//ublas::subrange(cpuAmatrix,numberV,numberV,numberV,numberV) = *initAMatrix;
	}
	
	cpuVectorType cpuBvector(numberV*2);
	cpuVectorType cpuXvector(numberV*2);
	
	gpuCompressedMatrixType gpuAmatrix(numberV*2,numberV*2);
	gpuVectorType gpuBvector(numberV*2);
	gpuVectorType gpuXvector(numberV*2);

	PolarList *nowp = NULL;
	for (int i = 0; i < numberV ; i++)
	{
		if(boundary[i]!=1)
		{
			cpuBvector(i) = 0.0f;
			cpuBvector(i+numberV) = 0.0f;
			if (initAMatrix == NULL)
			{
				nowp = PHead[i];			
				cpuAmatrix(i,i) = 1.0f;
				cpuAmatrix(numberV +i,numberV +i) = 1.0f;
				while(nextN(nowp)!=PTail[i])
				{
					nowp = nextN(nowp);						
					cpuAmatrix(i,nowp->ID) = -nowp->lambda;		
					cpuAmatrix(numberV +i,numberV + nowp->ID) = -nowp->lambda;
				}
			}
		}
		else
		{			
			//constraint 
			cpuBvector(i) = ioU[i];
			cpuBvector(i+numberV) = ioV[i];
			if (initAMatrix == NULL)
			{
				cpuAmatrix(i,i) = 1.0f;
				cpuAmatrix(numberV + i,numberV +i) = 1.0f;
			}
		}
	}
			
	
	
	if (!directsolver)
	{
		viennacl::copy(cpuAmatrix, gpuAmatrix);
		viennacl::copy(cpuBvector, gpuBvector);				
		viennacl::linalg::ilu0_tag ilu0_config;
		viennacl::linalg::ilu0_precond< gpuCompressedMatrixType > vcl_ilut(gpuAmatrix,ilu0_config);
		gpuXvector = viennacl::linalg::solve(gpuAmatrix,gpuBvector,viennacl::linalg::bicgstab_tag(error),vcl_ilut);
		//gpuXvector = viennacl::linalg::solve(gpuAmatrix,gpuBvector,viennacl::linalg::bicgstab_tag(error));
		viennacl::copy(gpuXvector, cpuXvector);
		
	}
	else
	{
		cpuDenseMatrixType denMatA_cpu = cpuAmatrix;
		gpuDenseMatrixType denMatA_gpu ;
		viennacl::copy(denMatA_cpu, denMatA_gpu);	
		viennacl::copy(cpuBvector, gpuBvector);	
	
		//boost::numeric::ublas::permutation_matrix<size_t> pm ( denMatA_cpu.size1() );
		//boost::numeric::ublas::lu_factorize( denMatA_cpu, pm );
		//boost::numeric::ublas::lu_substitute( denMatA_cpu, pm, cpuXvector );

		viennacl::linalg::lu_factorize(denMatA_gpu);
		viennacl::linalg::lu_substitute(denMatA_gpu,gpuBvector); //gpuBvector  got result
		viennacl::copy(gpuBvector, cpuXvector);
	}


	
	

	for(int i=0;i<numberV;i++)
	{
		if(boundary[i]!=1)
		{
			ioU[i] = cpuXvector(i);
			ioV[i] = cpuXvector(numberV + i);
					
		}
	}
	
	
	double *_sigma  = new double[numberV];	
	double candidate_l2 = GetStretchError(ioU,ioV);
	double previous_l2= 0;
	double *sigsum = new double[numberV];
	double *prevU = new double[numberV];
	double *prevV = new double[numberV];
	do
	{
		previous_l2 = candidate_l2;
		memcpy(prevU,ioU,sizeof(double)*numberV);
		memcpy(prevV,ioV,sizeof(double)*numberV);
		SetSigma(ioU,ioV,_sigma);
		for(int i=0;i<numberV;i++)
		{      
			if(boundary[i]!=1)
			{
				sigsum[i]=0.0;
				nowp = PHead[i];
				while(nextN(nowp)!=PTail[i])
				{
					nowp = nextN(nowp);
					//nowp->old_lambda = nowp->lambda;
					//nowp->lambda /= sigma[nowp->ID];
					
					sigsum[i] += (-cpuAmatrix(i,nowp->ID)/_sigma[nowp->ID]);
				}

				cpuAmatrix(i,i) = 1.0f;
				cpuAmatrix(i+numberV,i+numberV) = 1.0f;
				nowp = PHead[i];
				while(nextN(nowp)!=PTail[i])
				{
					nowp = nextN(nowp);
					double newLamda = ((cpuAmatrix(i,nowp->ID)/_sigma[nowp->ID]))/sigsum[i];
					cpuAmatrix(i,nowp->ID) = newLamda;	
					cpuAmatrix(i+numberV,nowp->ID+numberV) = newLamda;	
				}
			}
			else
			{
				cpuAmatrix(i,i) = 1.0f;
				cpuAmatrix(i+numberV,i+numberV) = 1.0f;
			}
		}

		if (!directsolver)
		{
			viennacl::copy(cpuAmatrix, gpuAmatrix);
			viennacl::copy(cpuBvector, gpuBvector);				
			viennacl::linalg::ilu0_tag ilu0_config;
			viennacl::linalg::ilu0_precond< gpuCompressedMatrixType > vcl_ilut(gpuAmatrix,ilu0_config);
			gpuXvector = viennacl::linalg::solve(gpuAmatrix,gpuBvector,viennacl::linalg::bicgstab_tag(error),vcl_ilut);
			//gpuXvector = viennacl::linalg::solve(gpuAmatrix,gpuBvector,viennacl::linalg::bicgstab_tag(error));
			viennacl::copy(gpuXvector, cpuXvector);
		
		}
		else
		{
			cpuDenseMatrixType denMatA_cpu = cpuAmatrix;
			gpuDenseMatrixType denMatA_gpu ;
			viennacl::copy(denMatA_cpu, denMatA_gpu);	
			viennacl::copy(cpuBvector, gpuBvector);	
			viennacl::linalg::lu_factorize(denMatA_gpu);
			viennacl::linalg::lu_substitute(denMatA_gpu,gpuBvector); //gpuBvector  got result
			viennacl::copy(gpuBvector, cpuXvector);
		}

		for(int i=0;i<numberV;i++)
		{
			if(boundary[i]!=1)
			{
				
				ioU[i] = cpuXvector(i);
				ioV[i] = cpuXvector(numberV + i);
					
			}
		}		
		candidate_l2 = GetStretchError(ioU,ioV);
	}
	while (previous_l2 > candidate_l2);

	memcpy(ioU,prevU,sizeof(double)*numberV);
	memcpy(ioV,prevV,sizeof(double)*numberV);

	delete [] _sigma;
	delete [] sigsum;
	delete [] prevU;
	delete [] prevV;
	return previous_l2;

}

double MyParameterization::ParametrizationOptimalGPU_2timeSolves(double *ioU,double *ioV,double error, cpuCompressedMatrixType *initAMatrix,bool directsolver,FILE* logFile)
{
	bool hasInitMatrix = false;
	cpuCompressedMatrixType cpuAmatrix(numberV,numberV);
	if (initAMatrix != NULL && 
		initAMatrix->size1() == initAMatrix->size2() &&
		initAMatrix->size1() == numberV)
	{
		cpuAmatrix = *initAMatrix;
		hasInitMatrix = true;
	}
	gpuCompressedMatrixType gpuAmatrix(numberV,numberV);	


	cpuVectorType cpuB1vector(numberV);
	cpuVectorType cpuB2vector(numberV);
	cpuVectorType cpuX1vector(numberV);	
	cpuVectorType cpuX2vector(numberV);	

	gpuVectorType gpuB1vector(numberV);
	gpuVectorType gpuB2vector(numberV);
	gpuVectorType gpuX1vector(numberV);	
	gpuVectorType gpuX2vector(numberV);	
	

	PolarList *nowp = NULL;
	for (int i = 0; i < numberV ; i++)
	{
		if(boundary[i]!=1)
		{
			cpuB1vector(i) = 0.0f;
			cpuB2vector(i) = 0.0f;
			if (!hasInitMatrix)
			{
				nowp = PHead[i];			
				cpuAmatrix(i,i) = 1.0f;
				
				while(nextN(nowp)!=PTail[i])
				{
					nowp = nextN(nowp);						
					cpuAmatrix(i,nowp->ID) = -nowp->lambda;	
				}
			}
		}
		else
		{			
			//constraint 
			cpuB1vector(i) = ioU[i];
			cpuB2vector(i) = ioV[i];			
			if (!hasInitMatrix)
			{
				cpuAmatrix(i,i) = 1.0f;
			}
		}
	}

	
	
	if (!directsolver)
	{
		viennacl::copy(cpuAmatrix, gpuAmatrix);
		viennacl::copy(cpuB1vector, gpuB1vector);
		viennacl::copy(cpuB2vector, gpuB2vector);
		
		viennacl::linalg::ilu0_tag ilu0_config;
		viennacl::linalg::ilu0_precond< gpuCompressedMatrixType > vcl_ilut(gpuAmatrix,ilu0_config);
		//viennacl::linalg::ichol0_tag ichol0_config;
		//viennacl::linalg::ichol0_precond< gpuCompressedMatrixType > vcl_ichol(gpuAmatrix,ichol0_config);
		
		gpuX1vector = viennacl::linalg::solve(gpuAmatrix,gpuB1vector,viennacl::linalg::bicgstab_tag(error),vcl_ilut);
		gpuX2vector = viennacl::linalg::solve(gpuAmatrix,gpuB2vector,viennacl::linalg::bicgstab_tag(error),vcl_ilut);
		//gpuX1vector = viennacl::linalg::solve(gpuAmatrix,gpuB1vector,viennacl::linalg::gmres_tag(error),vcl_ichol);
		//gpuX2vector = viennacl::linalg::solve(gpuAmatrix,gpuB2vector,viennacl::linalg::gmres_tag(error),vcl_ichol);
		
		viennacl::copy(gpuX1vector, cpuX1vector);
		viennacl::copy(gpuX2vector, cpuX2vector);
		
	}
	else
	{
		
		cpuDenseMatrixType denMatA_cpu( cpuAmatrix);
		gpuDenseMatrixType denMatA_gpu(numberV,numberV);
		viennacl::copy(denMatA_cpu, denMatA_gpu);
		viennacl::copy(cpuB1vector, gpuB1vector);
		viennacl::copy(cpuB2vector, gpuB2vector);

		viennacl::linalg::lu_factorize(denMatA_gpu);
		viennacl::linalg::lu_substitute(denMatA_gpu,gpuB1vector); //gpuB1vector  got result
		viennacl::linalg::lu_substitute(denMatA_gpu,gpuB2vector); //gpuB2vector  got result
		
		viennacl::copy(gpuB1vector, cpuX1vector);
		viennacl::copy(gpuB2vector, cpuX2vector);
		
	}


	
#if 1

	for(int i=0;i<numberV;i++)
	{
		if(boundary[i]!=1)
		{
			ioU[i] = cpuX1vector(i);
			ioV[i] = cpuX2vector(i);
		}
	}
	
	
	double *_sigma  = new double[numberV];	
	double candidate_l2 = GetStretchError(ioU,ioV);
	double previous_l2= 0;
	double *sigsum = new double[numberV];
	double *prevU = new double[numberV];
	double *prevV = new double[numberV];

	do
	{
		previous_l2 = candidate_l2;
		memcpy(prevU,ioU,sizeof(double)*numberV);
		memcpy(prevV,ioV,sizeof(double)*numberV);
		SetSigma(ioU,ioV,_sigma);
		for(int i=0;i<numberV;i++)
		{      
			if(boundary[i]!=1)
			{
				sigsum[i]=0.0;
				nowp = PHead[i];
				while(nextN(nowp)!=PTail[i])
				{
					nowp = nextN(nowp);
					//nowp->old_lambda = nowp->lambda;
					//nowp->lambda /= sigma[nowp->ID];
					
					sigsum[i] += (-cpuAmatrix(i,nowp->ID)/_sigma[nowp->ID]);
				}

				cpuAmatrix(i,i) = 1.0f;
				
				nowp = PHead[i];
				while(nextN(nowp)!=PTail[i])
				{
					nowp = nextN(nowp);
					double newLamda = ((cpuAmatrix(i,nowp->ID)/_sigma[nowp->ID]))/sigsum[i];
					cpuAmatrix(i,nowp->ID) = newLamda;
				}
			}
			else
			{
				cpuAmatrix(i,i) = 1.0f;
			}
		}

		if (!directsolver)
		{
			viennacl::copy(cpuAmatrix, gpuAmatrix);
			viennacl::copy(cpuB1vector, gpuB1vector);
			viennacl::copy(cpuB2vector, gpuB2vector);
		
			viennacl::linalg::ilu0_tag ilu0_config;
			viennacl::linalg::ilu0_precond< gpuCompressedMatrixType > vcl_ilut(gpuAmatrix,ilu0_config);
			gpuX1vector = viennacl::linalg::solve(gpuAmatrix,gpuB1vector,viennacl::linalg::bicgstab_tag(error),vcl_ilut);
			gpuX2vector = viennacl::linalg::solve(gpuAmatrix,gpuB2vector,viennacl::linalg::bicgstab_tag(error),vcl_ilut);
		
			viennacl::copy(gpuX1vector, cpuX1vector);
			viennacl::copy(gpuX2vector, cpuX2vector);
		
		}
		else
		{
			cpuDenseMatrixType denMatA_cpu( cpuAmatrix);
			gpuDenseMatrixType denMatA_gpu(numberV,numberV);
			viennacl::copy(denMatA_cpu, denMatA_gpu);
			viennacl::copy(cpuB1vector, gpuB1vector);
			viennacl::copy(cpuB2vector, gpuB2vector);

			viennacl::linalg::lu_factorize(denMatA_gpu);
			viennacl::linalg::lu_substitute(denMatA_gpu,gpuB1vector); //gpuB1vector  got result
			viennacl::linalg::lu_substitute(denMatA_gpu,gpuB2vector); //gpuB2vector  got result
		
			viennacl::copy(gpuB1vector, cpuX1vector);
			viennacl::copy(gpuB2vector, cpuX2vector);
		
			
		}

		for(int i=0;i<numberV;i++)
		{
			if(boundary[i]!=1)
			{
				ioU[i] = cpuX1vector(i);
				ioV[i] = cpuX2vector(i);
			}
		}		
		candidate_l2 = GetStretchError(ioU,ioV);
	}
	while (previous_l2 > candidate_l2);

	memcpy(ioU,prevU,sizeof(double)*numberV);
	memcpy(ioV,prevV,sizeof(double)*numberV);

	delete [] _sigma;
	delete [] sigsum;
	delete [] prevU;
	delete [] prevV;
	return previous_l2;
	#else
	return 0;
#endif
	
}

void MyParameterization::SetSigma(double *ipU,double *ipV,double *opSigma,double gamma )
{
	IDList *now=NULL;
	double varphi,ddv,dsize1,sumarea;
	double dddhval=0.0;
	double localsum=0.0;
	Point3d _bc[2];
	double *_E = new double[numberF];
	double *_G = new double[numberF];
	for(int i=0;i<numberF;i++)
	{    
		dsize1 = PT->getParametricA(ipV[Face[i][0]],
									ipV[Face[i][1]],
									ipV[Face[i][2]],
									ipU[Face[i][0]],
									ipU[Face[i][1]],
									ipU[Face[i][2]]);
		PT->setParametricDs(&_bc[0],
							point[Face[i][0]],point[Face[i][1]],point[Face[i][2]],
							ipV[Face[i][0]],ipV[Face[i][1]],ipV[Face[i][2]],dsize1);
		PT->setParametricDt(&_bc[1],
							point[Face[i][0]],point[Face[i][1]],point[Face[i][2]],
							ipU[Face[i][0]],ipU[Face[i][1]],ipU[Face[i][2]],dsize1);
    
		_E[i] = PT->InnerProduct(&_bc[0],&_bc[0]);
		_G[i] = PT->InnerProduct(&_bc[1],&_bc[1]);
	}

	for(int i=0;i<numberV;i++)
	{
         
		opSigma[i]=0.0;
		now = FHead[i];
		varphi=0.0;
		localsum=0.0;   
    
		while(next(now)!=FTail[i])
		{
			now = next(now);
			varphi += (areaMap3D[now->ID]*(0.5*(_E[now->ID]+_G[now->ID])));
			localsum += (areaMap3D[now->ID]);
		}
    
		opSigma[i] = sqrt((varphi/localsum));    
		opSigma[i] = pow(opSigma[i],gamma);  
	}
	delete [] _E;
	delete [] _G;
  
}

void ParametrizationOptimalGPU(Polyhedron *poly,FILE* logFile)
{
	int numberV = poly->numberV;
	//int numberV = 100;
	cpuCompressedMatrixType cpuAmatrix(numberV*2,numberV*2);
	cpuVectorType cpuBvector(numberV*2);
	cpuVectorType cpuXvector(numberV*2);
	
	gpuCompressedMatrixType gpuAmatrix(numberV*2,numberV*2);
	gpuVectorType gpuBvector(numberV*2);
	gpuVectorType gpuXvector(numberV*2);
	
	
	
	
	for(int i=0;i<numberV;i++)
	{		
		if(poly->boundary[i]!=1)
		{			
			cpuBvector(i) = 0.0f;
			cpuBvector(i+numberV) = 0.0f;			
		}
		else
		{
			//constraint 
			cpuBvector(i) = poly->pU[i];
			cpuBvector(i+numberV) = poly->pV[i];
		}
	}
	//setFloaterC();
	//poly->SortIndexP();

	PolarList *nowp = NULL;

	//for U
	nowp = NULL;
	for (int i = 0; i < numberV ; i++)
	{
		if(poly->boundary[i]!=1)
		{
			nowp = poly->PHead[i];
			
			cpuAmatrix(i,i) = 1.0f;
			while(nextN(nowp)!=poly->PTail[i])
			{
				nowp = nextN(nowp);						
				cpuAmatrix(i,nowp->ID) = -nowp->lambda;				
			}			
		}
		else
		{			
			cpuAmatrix(i,i) = 1.0f;
		}
	}
			
	//for V
	nowp = NULL;
	for (int i = 0; i < numberV ; i++)
	{
		if(poly->boundary[i]!=1)
		{
			nowp = poly->PHead[i];			
			cpuAmatrix(numberV +i,numberV +i) = 1.0f;			
			while(nextN(nowp)!=poly->PTail[i])
			{
				nowp = nextN(nowp);
				cpuAmatrix(numberV +i,numberV + nowp->ID) = -nowp->lambda;
				
			}			
		}
		else
		{
			cpuAmatrix(numberV + i,numberV +i) = 1.0f;				
		}
	}
	viennacl::copy(cpuAmatrix, gpuAmatrix);
	viennacl::copy(cpuBvector, gpuBvector);

	gpuXvector = viennacl::linalg::solve(gpuAmatrix,gpuBvector,viennacl::linalg::bicgstab_tag());
	viennacl::copy(gpuXvector, cpuXvector);

	for(int i=0;i<numberV;i++)
	{
		if(poly->boundary[i]!=1)
		{
			poly->pU[i] = cpuXvector(i);
			poly->pV[i] = cpuXvector(numberV + i);
					
		}
	}
	//double initialstrech = poly->getCurrentE();
	
	

#if 0

double *UaXY = new double[2*(numberV)+1];
    double *vecb = new double[2*(numberV)+1];
  
	int i;
	IDList *now;
	IDList *now2;
	PolarList *nowp;
	level=0;
  
	int nonzero=(numberV);
	for(i=0;i<numberV;i++)
	{
		vecb[i+1]=0.0;
		if(boundary[i]!=1)
		{
			nonzero += neighborI[i];			
		}
	}

	int iter=0;
	double linerr=0.0;
	double weight=0.0;
  
	PCBCGSolver *mybcg = new PCBCGSolver(2*nonzero);
	double *sigsum = new double[numberV];

	setFloaterC();
	/*
	if(weighttype==0)
	{
		setFloaterC();
	}
	else if(weighttype==1)
	{
		setLaplaceC();
	}
	else if(weighttype==2)
	{
		setEckHC();
	}
	else if(weighttype==3)
	{
		setDesbrunC();
	}
	else if(weighttype==4)
	{
		setMVCC();
	}
	else
	{
		setFloaterC();
	}
	*/
    
  
	SortIndexP();
  
  

	for(i=0;i<numberV;i++)
	{    
		if(boundary[i]!=1)
		{
			mybcg->sa[i+1] = 1.0;
			mybcg->sa[i+1+numberV] = 1.0;
			vecb[i+1] = 0.0;
			vecb[i+1+numberV] = 0.0;
		}
		else
		{
			mybcg->sa[i+1] = 1.0;
			vecb[i+1] = pU[i];
			mybcg->sa[i+1+numberV] = 1.0;
			vecb[i+1+numberV] = pV[i];
		}
	}

	mybcg->ija[1] = 2*(numberV)+2;
	int dlk=2*(numberV)+1;
  
	for(i=0;i<numberV;i++)
	{
		if(boundary[i]!=1)
		{
			nowp = PHead[i];      
			while(nextN(nowp)!=PTail[i])
			{
				nowp = nextN(nowp);
				++dlk;
				mybcg->sa[dlk] = -nowp->lambda;
				mybcg->ija[dlk]=nowp->ID+1;
			}
		}
		mybcg->ija[i+1+1]=dlk+1;
	}
	for(i=0;i<numberV;i++)
	{
		if(boundary[i]!=1)
		{
			nowp = PHead[i];
			while(nextN(nowp)!=PTail[i])
			{
				nowp = nextN(nowp);
				++dlk;
				mybcg->sa[dlk] = -nowp->lambda;
				mybcg->ija[dlk]=nowp->ID+numberV+1;
			}
		}
		mybcg->ija[i+numberV+1+1]=dlk+1;
	}
  
	for(i=0;i<numberV;i++)
	{
		UaXY[i+1] = pU[i];
		UaXY[i+numberV+1] = pV[i];
	}
	mybcg->linbcg(((unsigned long)(2*(numberV))),vecb,UaXY,1,error,itenum,&iter,&linerr);
  
	for(i=0;i<numberV;i++)
	{
		if(boundary[i]!=1)
		{
			pU[i] = UaXY[i+1];
			pV[i] = UaXY[i+numberV+1];
		}
	}
  
	// Re-solving linear system
	double initialstrech=0.0;
	double currentstrech=0.0;
	Point2d *prevU = new Point2d [numberV];  

	initialstrech = getCurrentE();
	int kk = 0;  

	printf("U%d  STRETCH: %f\n",kk,initialstrech); //u0
	if (logFile)
	{
		fprintf(logFile,"U%d  STRETCH: %f\n",kk,initialstrech); //u0
	}
	for(kk=0;kk<itenum;kk++)
	{
		setSigmaZero();
		for(i=0;i<numberV;i++)
		{
      
			if(boundary[i]!=1)
			{
				sigsum[i]=0.0;
				nowp = PHead[i];
				while(nextN(nowp)!=PTail[i])
				{
					nowp = nextN(nowp);
					nowp->old_lambda = nowp->lambda;
					nowp->lambda /= sigma[nowp->ID];
					sigsum[i] += nowp->lambda;
				}
				mybcg->sa[i+1] = 1.0;
				mybcg->sa[i+1+numberV] = 1.0;
			}
			else
			{
				mybcg->sa[i+1] = 1.0;
				mybcg->sa[i+1+numberV] = 1.0;
			}
		}
		dlk=2*(numberV)+1;
    
		for(i=0;i<numberV;i++)
		{
			if(boundary[i]!=1)
			{
				nowp = PHead[i];
	
				while(nextN(nowp)!=PTail[i])
				{
					nowp = nextN(nowp);
					++dlk;
					mybcg->sa[dlk] = -nowp->lambda/sigsum[i];
				}
			}
		}

		for(i=0;i<numberV;i++)
		{
			if(boundary[i]!=1)
			{
				nowp = PHead[i];
				while(nextN(nowp)!=PTail[i])
				{
					nowp = nextN(nowp);
					++dlk;
					mybcg->sa[dlk] = -nowp->lambda/sigsum[i];
				}
			}
		}
     
		for(i=0;i<numberV;i++)
		{
			UaXY[i+1] = pU[i];
			UaXY[i+numberV+1] = pV[i];
		}

		mybcg->linbcg(((unsigned long)(2*(numberV))),vecb,UaXY,1,error,itenum,&iter,&linerr); 
		for(i=0;i<numberV;i++)
		{
			if(boundary[i]!=1)
			{
				prevU[i].x = pU[i];
				prevU[i].y = pV[i];
	
				pU[i] = UaXY[i+1];
				pV[i] = UaXY[i+numberV+1];
			}
		}
		currentstrech = getCurrentE();
		printf("currentstrech U%d= %lf\n",kk+1,currentstrech);
		printf("U%d  STRETCH: %f\n",kk+1,currentstrech); 

		if (logFile)
		{
			fprintf(logFile,"U%d  STRETCH: %f\n",kk+1,currentstrech); 
		}

		if(initialstrech<currentstrech)
		{
			for(i=0;i<numberV;i++)
			{
				if(boundary[i]!=1)
				{
					pU[i] = prevU[i].x;
					pV[i] = prevU[i].y;
					
				}
			}

			resultStretch = initialstrech;
			break;		
		}
		else
		{
			initialstrech = currentstrech;
		}
	}
  
	level = kk;
 
 
	printf("STL2 (U%d) error = %lf\n", level ,resultStretch);
	delete mybcg;
	/*
	for(i=0;i<numberV;i++)
	{
		if(boundary[i]!=1)
		{
			IDtool->CleanNeighborPolar(PHead[i],PTail[i]);
      
			PHead[i] = new PolarList();
			PTail[i] = new PolarList();
			PHead[i]->next = PTail[i];
			PTail[i]->back = PHead[i];
		}
	}
	*/

	delete [] prevU;
	delete [] vecb;
	delete [] UaXY;
	delete [] sigsum;
#endif
}




double    MyParameterization::PARAM_PARALLEL_GPU(	PolarVertex *pIPV,
												int num_PV,
												FILE* logFile)
{
	

	
	iteNum = (pow((double)((numberV/20000) + 1),2)) *2000;	
	if (pU)
	{
		delete [] pU;
		pU = NULL;
	}
	if (pV)
	{
		delete [] pV;
		pV = NULL;
	}


    boundarytype=0;
	setPolarMap();
	IDList *BpointH = new IDList();
	IDList *BpointT = new IDList();
    
	BpointH->next = BpointT;
	BpointT->back = BpointH;
	
	double tlength=0.0;
	int numBorderPoint = 0;
	CalBorderPath(BpointH,BpointT,&tlength,&numBorderPoint);

	IDList *now = BpointH;
	


	double loop = 0;

	double bestStretch = DBL_MAX;
	int best_startID = -1;	
	int worst_startID = -1;
	double worstStretch = -1;
	


	//ResetInnerLambda();
	setFloaterC();
	SortIndexP();
	//record result as array.
	vector<int> bottomLeftVertexIDList;
	//double *stretch_each_corner = new double[numBorderPoint];

	printf("=== NUMBER BORDER EDGES : %d ===\n",numBorderPoint);
	if (logFile)
		fprintf(logFile,"=== NUMBER BORDER EDGES : %d ===\n",numBorderPoint);
	while (loop < tlength*0.25 && next(now) != BpointT)
	{
		now = next(now);		
		loop += PT->Distance(point[now->ID],point[next(now)->ID]);
		bottomLeftVertexIDList.push_back(now->ID);
	}

	//create initial A matrix  ,it is same for all condition
	//to boost up speed
	/*
	cpuCompressedMatrixType initialAMatrix(numberV*2,numberV*2);
	for (int i = 0; i < numberV ; i++)
	{
		if(boundary[i]!=1)
		{
			
			PolarList *nowp = PHead[i];			
			initialAMatrix(i,i) = 1.0f;
			initialAMatrix(numberV +i,numberV +i) = 1.0f;
			while(nextN(nowp)!=PTail[i])
			{
				nowp = nextN(nowp);						
				initialAMatrix(i,nowp->ID) = -nowp->lambda;		
				initialAMatrix(numberV +i,numberV + nowp->ID) = -nowp->lambda;
			}			
		}
		else
		{			
			//constraint
			initialAMatrix(i,i) = 1.0f;
			initialAMatrix(numberV + i,numberV +i) = 1.0f;
		}
	}
	*/
	
	cpuCompressedMatrixType initialAMatrix(numberV,numberV);
	for (int i = 0; i < numberV ; i++)
	{
		if(boundary[i]!=1)
		{
			
			PolarList *nowp = PHead[i];			
			initialAMatrix(i,i) = 1.0f;			
			while(nextN(nowp)!=PTail[i])
			{
				nowp = nextN(nowp);						
				initialAMatrix(i,nowp->ID) = -nowp->lambda;						
			}			
		}
		else
		{			
			//constraint
			initialAMatrix(i,i) = 1.0f;
		}
	}
	
	double *bestU = NULL;
	double *bestV = NULL;
	std::vector<double *>_resultU(bottomLeftVertexIDList.size(),NULL);
	std::vector<double *>_resultV(bottomLeftVertexIDList.size(),NULL);
	std::vector<double>_resultError(bottomLeftVertexIDList.size(),0.0);
	
	unsigned int n = bottomLeftVertexIDList.size(); 
#if 1
	
	concurrency::SchedulerPolicy oldpolicy = concurrency::CurrentScheduler::GetPolicy();	
	concurrency::SchedulerPolicy policy(oldpolicy);
	if (policy.GetPolicyValue(concurrency::MaxConcurrency) > 10)
		policy.SetConcurrencyLimits(1,10);	
	
	concurrency::CurrentScheduler::Create(policy);	
	concurrency::parallel_for(0u, n, [&_resultU,&_resultV,&_resultError,&initialAMatrix,bottomLeftVertexIDList,BpointH,BpointT,tlength,this](unsigned int i)  
	{
		SquareParametrizationGPU(bottomLeftVertexIDList[i], BpointH,BpointT, tlength,&initialAMatrix,_resultU[i],_resultV[i],_resultError[i],true,false);		
	});
	concurrency::CurrentScheduler::Detach();
	
	for (int i = 0 ; i < bottomLeftVertexIDList.size(); i++)
	{
		if (_resultError[i] < bestStretch )
		{
			if (bestU)
				delete [] bestU;
			if (bestV)
				delete [] bestV;
			bestU = _resultU[i];
			bestV = _resultV[i];
			
			bestStretch = _resultError[i];
			best_startID = i;
		}
		else
		{
			
			delete [] _resultU[i];
			delete [] _resultV[i];
		}

		if (_resultError[i] > worstStretch)
		{
			worstStretch = _resultError[i];
			worst_startID = i;
		}	
	}
	
#else
	
	for (int i = 0 ; i < bottomLeftVertexIDList.size(); i++)
	{
		double *resultU = NULL;
		double *resultV = NULL;
		double resultErr  = 0;
		SquareParametrizationGPU (bottomLeftVertexIDList[i], BpointH,BpointT, tlength,&initialAMatrix,resultU,resultV,resultErr,false,false);
		if (resultErr < bestStretch )
		{
			if (bestU)
				delete [] bestU;
			if (bestV)
				delete [] bestV;
			bestU = resultU;
			bestV = resultV;
			
			bestStretch = resultErr;
			best_startID = i;
		}
		else
		{
			
			delete [] resultU;
			delete [] resultV;
		}

		if (resultErr > worstStretch)
		{
			worstStretch = resultErr;
			worst_startID = i;
		}		
		
	}
	
#endif	
	for (int i=0;i<numberV;i++)
	{				
		pIPV[i].u = bestU[i];
		pIPV[i].v = bestV[i];
	}
	double *resultCircleU = NULL;
	double *resultCircleV = NULL;
	double resultCircleErr = 0;
	CircleParametrizationGPU(BpointH,BpointT, tlength,&initialAMatrix,resultCircleU,resultCircleV,resultCircleErr,false,false);


	if (bestU)
		delete [] bestU;
	if (bestV)
		delete [] bestV;

	if (resultCircleU)
		delete [] resultCircleU;
	if (resultCircleV)
		delete [] resultCircleV;
	
	printf("=== Find best stretch  of TEST%d  (best corner at %f) ===\n",best_startID,bestStretch);
	if (logFile)
		fprintf(logFile,"=== Find best stretch of TEST%d (best corner ERR = %f)===\n",best_startID,bestStretch);
	
	printf("=== Find worst stretch  of TEST%d  (worst corner at %f) ===\n",worst_startID,worstStretch);
	if (logFile)
		fprintf(logFile,"=== Find worst stretch  of TEST%d  (worst corner ERR = %f) ===\n",worst_startID,worstStretch);

	printf("=== Circular Parameterization  stretch = %f ===\n",resultCircleErr);
	if (logFile)
		fprintf(logFile,"=== Circular Parameterization  stretch = %f ===\n",resultCircleErr);

		
	IDtool->CleanNeighbor(BpointH,BpointT);
	
	this->resultStretch = bestStretch;
	return bestStretch;	
}